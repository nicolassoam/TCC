#include "hip/hip_runtime.h"
#include "gpu.cuh"

namespace GPU
{
    void cleanupDeviceData(DeviceDataManager& d_manager) 
    {
        hipFree(d_manager.d_aircraftTypes);
        hipFree(d_manager.d_routes);
        hipFree(d_manager.d_allTicketPrices);
        hipFree(d_manager.d_allCaskValues);
        hipFree(d_manager.d_allDemands);

        d_manager.d_aircraftTypes = nullptr;
        d_manager.d_routes = nullptr;
        d_manager.d_allTicketPrices = nullptr;
        d_manager.d_allCaskValues = nullptr;
        d_manager.d_allDemands = nullptr;
        d_manager.numAircraftTypes = 0;
        d_manager.numTimeWindows = 0;
        d_manager.numRoutes = 0;
    };

    void cleanupDevicePopulation(DevicePopulationManager& d_manager) 
    {
        hipFree(d_manager.d_individuals);
        hipFree(d_manager.d_allFlights);
        hipFree(d_manager.d_allAllowedAircraft);

        d_manager.d_individuals = nullptr;
        d_manager.d_allFlights = nullptr;
        d_manager.d_allAllowedAircraft = nullptr;
    }
    void setupDevicePopulation(
        DevicePopulationManager& d_manager,
        const std::vector<Individual>& h_population)
    {
        if (h_population.empty()) {
            std::cerr << "Warning: Host population is empty." << std::endl;
            return;
        }

        d_manager.population_size = h_population.size();

        try {

            std::vector<GPUIndividual> h_gpuIndividuals;
            std::vector<GPUFlight>     h_allFlights;
            std::vector<char>          h_allAllowedAircraft;

            int currentFlightOffset = 0;
            int currentMaskOffset = 0;

            for (const auto& cpuInd : h_population) 
            {
                GPUIndividual gpuIndMeta;
                gpuIndMeta.fitness = cpuInd.fitness;

                gpuIndMeta.scheduleOffset = currentFlightOffset;
                gpuIndMeta.scheduleSize = cpuInd.schedule.size();

                gpuIndMeta.allowedAircraftOffset = currentMaskOffset;

                h_gpuIndividuals.push_back(gpuIndMeta);

                h_allFlights.insert(h_allFlights.end(), cpuInd.schedule.begin(), cpuInd.schedule.end());
                h_allAllowedAircraft.insert(h_allAllowedAircraft.end(), cpuInd.allowedAircraft.begin(), cpuInd.allowedAircraft.end());

                currentFlightOffset += cpuInd.schedule.size();
                currentMaskOffset += cpuInd.allowedAircraft.size();
            }

            d_manager.totalFlights = h_allFlights.size();

            // ========================================================================
            // Part B: Allocate memory on the GPU for the flattened arrays
            // ========================================================================

            cudaCheck(hipMalloc(&d_manager.d_individuals, sizeof(GPUIndividual) * h_gpuIndividuals.size()));
            cudaCheck(hipMalloc(&d_manager.d_allFlights, sizeof(GPUFlight) * h_allFlights.size()));
            cudaCheck(hipMalloc(&d_manager.d_allAllowedAircraft, sizeof(char) * h_allAllowedAircraft.size()));

            // ========================================================================
            // Part C: Copy the flattened host vectors to the device
            // ========================================================================

            cudaCheck(hipMemcpy(d_manager.d_individuals, h_gpuIndividuals.data(), sizeof(GPUIndividual) * h_gpuIndividuals.size(), hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_manager.d_allFlights, h_allFlights.data(), sizeof(GPUFlight) * h_allFlights.size(), hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_manager.d_allAllowedAircraft, h_allAllowedAircraft.data(), sizeof(char) * h_allAllowedAircraft.size(), hipMemcpyHostToDevice));

        }
        catch (const std::exception& e) 
        {
            std::cerr << e.what() << std::endl;
        }
    }

    void setupDeviceData(DeviceDataManager& d_manager, const std::vector<AircraftType>& h_aircraftTypes,
        const std::vector<Route>& h_routes,int numTimeWindows)
    {
        d_manager.numAircraftTypes = h_aircraftTypes.size();
        d_manager.numRoutes = h_routes.size();
        d_manager.numTimeWindows = numTimeWindows;

        try {

            // 1. Flatten AircraftType data
            std::vector<GPUAircraftType> h_gpuAircraftTypes;
            for (const auto& ac : h_aircraftTypes) 
            {
                h_gpuAircraftTypes.push_back({ ac.id, ac.capacity, ac.rangeKM });
            }

            // 2. Flatten all sub-arrays (prices, casks, demands) into single large vectors
            std::vector<double> h_allTicketPrices;
            std::vector<double> h_allCaskValues;
            std::vector<int>    h_allDemands;

            for (const auto& route : h_routes) 
            {
                for (int i = 0; i < d_manager.numAircraftTypes; ++i) {
                    h_allTicketPrices.push_back(route.ticketPrices.at(i));
                    h_allCaskValues.push_back(route.caskValues.at(i));
                }
                h_allDemands.insert(h_allDemands.end(), route.demandPerWindow.begin(), route.demandPerWindow.end());
            }

            // ========================================================================
            // Part B: Allocate memory on the GPU
            // ========================================================================

            cudaCheck(hipMalloc(&d_manager.d_aircraftTypes, sizeof(GPUAircraftType) * d_manager.numAircraftTypes));
            cudaCheck(hipMalloc(&d_manager.d_routes, sizeof(GPURoute) * d_manager.numRoutes));

            cudaCheck(hipMalloc(&d_manager.d_allTicketPrices, sizeof(double) * h_allTicketPrices.size()));
            cudaCheck(hipMalloc(&d_manager.d_allCaskValues, sizeof(double) * h_allCaskValues.size()));
            cudaCheck(hipMalloc(&d_manager.d_allDemands, sizeof(int) * h_allDemands.size()));


            cudaCheck(hipMemcpy(d_manager.d_aircraftTypes, h_gpuAircraftTypes.data(), sizeof(GPUAircraftType) * d_manager.numAircraftTypes, hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_manager.d_allTicketPrices, h_allTicketPrices.data(), sizeof(double) * h_allTicketPrices.size(), hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_manager.d_allCaskValues, h_allCaskValues.data(), sizeof(double) * h_allCaskValues.size(), hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(d_manager.d_allDemands, h_allDemands.data(), sizeof(int) * h_allDemands.size(), hipMemcpyHostToDevice));


            std::vector<GPURoute> h_gpuRoutes;
            for (int i = 0; i < d_manager.numRoutes; ++i) 
            {
                GPURoute tempRoute;
                tempRoute.id = h_routes[i].id;
                tempRoute.originId = h_routes[i].originId;
                tempRoute.destinationId = h_routes[i].destinationId;
                tempRoute.distanceKM = h_routes[i].distanceKM;

                tempRoute.ticketPrices = d_manager.d_allTicketPrices + (i * d_manager.numAircraftTypes);
                tempRoute.caskValues = d_manager.d_allCaskValues + (i * d_manager.numAircraftTypes);
                tempRoute.demandPerWindow = d_manager.d_allDemands + (i * d_manager.numTimeWindows);

                h_gpuRoutes.push_back(tempRoute);
            }

            cudaCheck(hipMemcpy(d_manager.d_routes, h_gpuRoutes.data(), sizeof(GPURoute) * d_manager.numRoutes, hipMemcpyHostToDevice));

        }
        catch (const std::exception& e) {
            std::cerr << e.what() << std::endl;
        }
    }


    __global__ void evaluateFitnessKernel(
        GPUIndividual* d_individuals,
        GPUFlight* d_allFlights,
        char* d_allAllowedAircraft,
        int population_size,

        GPURoute* d_routes,
        GPUAircraftType* d_aircraftTypes,
        int numAircraftTypes,
        int routeOffset,
        int max_assigned_types)
    {
        int individual_idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (individual_idx >= population_size) 
        {
            return;
        }

        GPUIndividual myIndMeta = d_individuals[individual_idx];
        GPUFlight* mySchedule = d_allFlights + myIndMeta.scheduleOffset;
        char* myAllowedMask = d_allAllowedAircraft + myIndMeta.allowedAircraftOffset;

        int allowedCount = 0;
        for (int i = 0; i < numAircraftTypes; ++i) 
        {
            if (myAllowedMask[i]) {
                allowedCount++;
            }
        }

        if (allowedCount == 0 || allowedCount > max_assigned_types) 
        {
            d_individuals[individual_idx].fitness = -1.0e18f; 
            return;
        }

        double totalProfit = 0.0; 
        for (int i = 0; i < myIndMeta.scheduleSize; ++i) 
        {
            GPUFlight flight = mySchedule[i];

            int returnRouteId = flight.outboundRouteId + routeOffset;

            const GPURoute& routeOut = d_routes[flight.outboundRouteId];
            const GPURoute& routeIn = d_routes[returnRouteId];
            const GPUAircraftType& aircraft = d_aircraftTypes[flight.aircraftTypeId];

            double priceOut = routeOut.ticketPrices[aircraft.id];
            double priceIn = routeIn.ticketPrices[aircraft.id];
            double caskOut = routeOut.caskValues[aircraft.id];
            double caskIn = routeIn.caskValues[aircraft.id];

            double outboundRevenue = flight.outboundPassengersPerFlight * priceOut;
            double returnRevenue = flight.returnPassengersPerFlight * priceIn;
            double outboundCost = routeOut.distanceKM * aircraft.capacity * caskOut * flight.frequency;
            double returnCost = routeIn.distanceKM * aircraft.capacity * caskIn * flight.frequency;

            double revenue = outboundRevenue + returnRevenue;
            double cost = (outboundCost + returnCost);

            double profitPerTrip = revenue - cost;
            totalProfit += profitPerTrip;
        }

        d_individuals[individual_idx].fitness = static_cast<float>(totalProfit);
    }


    void kernelCaller(DeviceDataManager deviceData, Population& population, int currentGen)
    {
        DevicePopulationManager devicePopulationManager;
        std::chrono::duration<double> elapsed;

        setupDevicePopulation(devicePopulationManager, population);

        int threadsPerBlock = 128;
        hipDeviceProp_t props;
        int deviceId;
        hipGetDevice(&deviceId);
        hipGetDeviceProperties(&props, deviceId);
        int numSMs = props.multiProcessorCount;
        int blocksPerGrid = numSMs * 4;

        evaluateFitnessKernel << <blocksPerGrid, threadsPerBlock >> > (
            devicePopulationManager.d_individuals,
            devicePopulationManager.d_allFlights,
            devicePopulationManager.d_allAllowedAircraft,
            devicePopulationManager.population_size,
          
            deviceData.d_routes,
            deviceData.d_aircraftTypes,
            deviceData.numAircraftTypes,
            10, 
            MAX_ASSIGNED_TYPES   
            );
        
        
        cudaCheck(hipGetLastError());
        cudaCheck(hipDeviceSynchronize());
        std::vector<GPUIndividual> h_gpuIndividualsResult(devicePopulationManager.population_size);

        // Perform the copy from Device memory to Host memory

        cudaCheck(hipMemcpy(
            h_gpuIndividualsResult.data(),                 
            devicePopulationManager.d_individuals,                 
            sizeof(GPUIndividual) * devicePopulationManager.population_size, 
            hipMemcpyDeviceToHost                           
        ));

        for (int i = 0; i < population.size(); ++i) 
        {
            population[i].fitness = h_gpuIndividualsResult[i].fitness;
        }

        //cleanupDeviceData(deviceData);
        cleanupDevicePopulation(devicePopulationManager);
    };
}